#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int main() {
    // 定义矩阵大小
    int m = 3, n = 4, k = 2;

    // 定义 alpha 和 beta
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // 创建和初始化矩阵 A, B 和 C
    // float A[m * k] = {
    //                     1, 2,
    //                     4, 5,
    //                     7, 8
    //                  };
    // float B[k * n] = {
    //                     1, 1, 1, 1,
    //                     1, 1, 1, 1
    //                  };

    // 创建和初始化矩阵 A, B 和 C
    float A[k * n] = {
                        1, 1, 1, 1,
                        1, 1, 1, 1
                     };

    float B[m * k] = {
                        1, 2,
                        4, 5,
                        7, 8
                     };

    float C[m * n] = {0};

    float *d_A, *d_B, *d_C;

    // 初始化 CUDA 和 cuBLAS
    hipMalloc((void **)&d_A, k * n * sizeof(float));
    hipMalloc((void **)&d_B, m * k * sizeof(float));
    hipMalloc((void **)&d_C, m * n * sizeof(float));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // 将矩阵数据复制到设备上
    hipMemcpy(d_A, A, k * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, m * k * sizeof(float), hipMemcpyHostToDevice);

    // 执行矩阵乘法
    hipblasGemmEx(handle,
                 HIPBLAS_OP_N, HIPBLAS_OP_N,
                 n, m, k,
                 &alpha,
                 d_A, HIP_R_32F, n,
                 d_B, HIP_R_32F, k,
                 &beta,
                 d_C, HIP_R_32F, n,
                 HIP_R_32F, HIPBLAS_GEMM_DEFAULT);

    // 将结果复制回主机
    hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // 打印结果
    printf("Result matrix C:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%f ", C[i * n + j]);
        }
        printf("\n");
    }

    // 清理资源
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

    return 0;
}
